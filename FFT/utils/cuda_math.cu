/*
 * Cuda Device Functions for Math Operations
 *
 */

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>


// Power Function for Complex
__device__ hipDoubleComplex pow_cuDoubleComplex(hipDoubleComplex z, int n){
    double r = hipCabs(z); // Magnitude
    double theta = atan2(hipCimag(z), hipCreal(z)); // Argument

    double rn = pow(r, n);
    double nTheta = n * theta;

    return make_hipDoubleComplex(rn * cos(nTheta), rn * sin(nTheta));
}
