/*
 * Cuda Debug Utilities
 */

#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>

void last_cuda_error(std::string event)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
        fprintf(stderr, "CUDA Error at %s: %s\n", event.c_str(), hipGetErrorString(err));
}
