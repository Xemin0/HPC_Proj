#include "hip/hip_runtime.h"
/*
 * 1D FFT Optimized with CUDA
 *
 * - Kernels
 * - Kernel Launching Methods
 * - 1D FFT_CUDA Wrapper Function
 */

#include <hip/hip_runtime.h>
#include <cmath>

#include "../lib/iterative_CT.h"

#define BLOCK_SIZE 64 // *** Subject to Change
#define WARP_SIZE 32
#define FULL_MASK 0xffffffff

typedef std::complex<double> Complex;
const double PI = 3.14159265358973238460;

// Kernel for BitReverse
// *** If it's worth to define bitReverse as a kernel ?? *** //
// *** parallelization may not be worthy for small data size *** //
// *** as the overhead of creating threads will outweigh the performance improvement *** //
// *** and Coalesced Memory Accesses may not be guaranteed ** //
__global__ void bitReverse_kernel(Complex *d_x, int N){
    /*
     * (Effectively?) Reverse the Bits of a Vector
     *
     *  - N: length of the vector x; a power of 2
     * 
     * ### Warp Level __shfl_sync may not be efficient ### 
     * Only (N/2 - 1) elements needed to initiate the swaps???
     * since each elememt is only swapped once
     */
    unsigned int startIdx = blockIdx.x * blockDim.x + threadIdx.x + 1; // Skipping the first element
    unsigned int stride = blockDim.x * gridDim.x;

    // Tiling
    // Less likely, but just in case the total number of threads = stride < N
    for (int i = startIdx; i < N - 1; i += stride){
        int j = 0;
        for (int k = 0; k < log2(N); ++k) // for each bit
            if (i & (1 << k))
                j |= (N >> (k+1));

        if (i < j){ // Swap elements at i and j
            Complex tmp = d_x[i];
            d_x[i] = d_x[j];
            d_x[j] = tmp;
        }
    }
}

// The Corresponding Kernel Launching Method


// ###########################################

__global__ void fft1d_kernel(Complex *d_x, int N){
    /*
     * 1 D FFT Kernel (** Currently Require blockDim.x = N/2 **)
     * 
     * stages : iterations for different len 
     * segment: at each stage, the vector is segmented into parts of size len
     *          each segment is handled by len/2 threads
     *          each thread is handling each butterfly pair 
     */
    // ###
    // Bit Reverse Part
    // ###
    unsigned int startIdx = blockIdx.x * blockDim.x + threadIdx.x + 1; // skipping the first element
    unsigned int stride = blockDim.x * gridDim.x;

    // Tiling
    // less likely, but just in case the total number of threads = stride < N
    for (int i = stratIdx; i < N - 1; i += stride){
        int j = 0;
        for (int k = 0; k < log2(N); ++k) // for each bit
            if (i & (1 << k))
                j |= (N >> (k+1));

        if (i < j){ // Swap element at i and j
            Complex tmp = d_x[i];
            d_x[i] = d_x[j];
            d_x[j] = tmp;
        }
    }
    // ###
    // Butterfly Step (Reworked from the Original Iterative Method)
    // ###

    /*
     * Assign len/2 threads to handle len/2 pairs of butterfly operations for each segment
     * Memory Access Pattern: Because the whole vector is accessed for each stage
     * 1. Load vector into a shared memory (**Require blockDim.x >= N** otherwise exchanging data between blocks would further introduce overheads) Shared memory limitation: 48kb or 96kb while Complex is of 16bytes
     * 2. Warp-Level Primitive `__shfl_sync()` because at each stage len is different, managing warps would be difficult
     * 3. A combination of the above two
     */
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // ### blockDim.x = N/2 with one Block

    __shared__ Complex x_shared0[N/2]; // shared memory for d_x First Half
    __shared__ Complex x_shared1[N/2]; // shared memory for d_x Second Half

    // Load d_x to be reused for each stage
    if (idx < N / 2){
        x_shared0[idx] = d_x[idx]; 
        x_shared1[idx] = d_x[idx + N/2];
    }

    __syncthreads();

    // Each Stage
    for (int len = 2; len <= N; len <<= 1){
        double angle = -2 * PI / len;
        Complex wlen(cos(angle), sin(angle));

        if (idx < N/2){ // boundary check for each thread ** half of threads are idling
            int segment_idx = idx / (len/2); // len/2 threads for each segment
            int local_tid = idx % (len/2);
            int segment_start = segment_idx * len;

            Complex w = pow(wlen, local_tid);

            int u_idx = segment_start + local_tid;
            int v_idx = u_idx + len/2;

            Complex u = (u_idx < N/2) ? x_shared0[u_idx] : x_shared1[u_idx - N/2];
            Complex v = (v_idx < N/2) ? x_shared0[v_idx] : x_shared1[v_idx - N/2];

            if (u_idx < N/2)
                x_shared0[u_idx] = u + v;
            else
                x_shared1[u_idx - N/2] = u + v;

            if (v_idx < N/2)
                x_shared0[v_idx] = u - v;
            else
                x_shared1[v_idx] = u - v;
        }
    }

    __syncthreads();

    // Write back to Global Memory
    if (idx < N / 2){
        d_x[idx] = x_shared0[idx];
        d_x[idx + N/2] = x_shared1[idx];
    }


void fft1d_device(Complex *d_x, int N)
{
    /*
     * Kernel Launching Method of fft_1d_kernel
     * both the input and the output stay on DEVICE
     * 
     * As the Butterfly Step requires N/2 threads
     * we will fix BlockDim = N/2 and GridDim = 1
     */
    dim3 nthreads(N/2, 1, 1);   // BlockDim
    dim3 nblocks(1, 1, 1);      // GridDim

    fft1d_kernel <<< nblocks, nthreads, 0, 0 >>> (d_x, N);
}


void fft1d_cu(Complex *h_x, int N)
{
    /*
     * Wrapper Function of 1D FFT with CUDA
     *
     * Assumes Data Allocated on HOST 
     * 1. Transfer Data to DEVICE
     * 2. Launch the Kernel
     * 3. Copy Back the Results
     */
    // Allocate memory on DEVICE
    Complex *d_x;
    hipMalloc( (void**) &d_x, sizeof(Complex)*N );

    // Copy the vector from HOST to DEVICE
    hipMemcpy(d_x, h_x, sizeof(Complex) * N, hipMemcpyHostToDevice);

    // Launch the Kernel
    fft1d_device(d_x, N);

    // Copy back the result from DEVICE to HOST
    hipMemcpy(h_x, d_x, sizeof(Complex) * N, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_x);
}

// ###########################################



